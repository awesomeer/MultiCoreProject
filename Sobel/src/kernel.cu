#include "hip/hip_runtime.h"
#include "../inc/kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <stdio.h>


/*
* This code is configured for a GTX 1060M with 1024 CUDA cores
*/


#define HEIGHT 720
#define WIDTH 1280
#define SIZE (3 * WIDTH * HEIGHT)


__managed__ char GX[9] = { 1, 0, -1,
						  2, 0, -2,
							1, 0, -1 };
__managed__ char GY[9] = { 1, 2, 1,
				   0, 0, 0,
				  -1,-2,-1 };

// __managed__ char gaussian_kernel[9] = { 
// 	1, 2, 1,
// 	2, 4, 2,
// 	1, 2, 1,
// };

__managed__ char gaussian_kernel[25] = { 
	1, 4, 6, 4, 1,
	4, 16, 24, 16, 4,
	6, 24, 36, 24, 6,
	4, 16, 24, 16, 4,
	1, 4, 6, 4, 1
};


// __managed__ char gaussian_kernel[9] = { 
// 	0, -1, 0,
// 	-1, 5, -1,
// 	0, -1, 0,
// };


unsigned char* greyScaleBuffer;
int *sobel; //1280x720
unsigned char *gaussian;
unsigned char *finished; //1280x720*3


__global__
void greyScale(unsigned char * frame, unsigned char*greyBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;
	int sum = (frame[3*index] + frame[3*index + 1] + frame[3*index + 2]) / 3;
	greyBuffer[index] = sum;

	//frame[3*index] = sum;
	//frame[3*index+1] = sum;
	//frame[3*index+2] = sum;
}


__device__
int index(int x, int y) {
	if (x >= WIDTH || y >= HEIGHT || x < 0 || y < 0)
		return -1;

	return x + y * WIDTH;
}

__device__ __forceinline__
int wrap(int val, int limit) {
	if (val < 0)
		return limit - 1;
	return val % limit;
}

__global__
void sobelOp(unsigned char * greyBuffer, int * sobelBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int xDir = 0;
	int yDir = 0;

	for (int r = -1; r < 2; r++) {
		int row = y + r;
		row = wrap(row, HEIGHT);

		for (int c = -1; c < 2; c++) {
			int col = x + c;
			col = wrap(col, WIDTH);

			xDir += greyBuffer[index(col, row)] * GX[(1 - c) + (1 - r) * 3];
			yDir += greyBuffer[index(col, row)] * GY[(1 - c) + (1 - r) * 3];
		}
	}

	__syncthreads();

	sobelBuffer[2*index(x, y)] = xDir;
	sobelBuffer[2*index(x, y)+1] = yDir;

}

__global__ void gaussian_filter(const unsigned char *gaussian_input, unsigned char *gaussian_output) {

    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

    if(row < HEIGHT && col < WIDTH) {
        int blur = 0;
        for(int i = -2; i < 3; i++) {
            for(int j = -2; j < 3; j++) {

                const unsigned int y = max(0, min(HEIGHT - 1, row + i));
                const unsigned int x = max(0, min(WIDTH - 1, col + j));

                char w = gaussian_kernel[(2-j) + (2-i) * 3];
				//printf("%f\n", w);
                blur += w * gaussian_input[x + y * WIDTH];
            }
        }
		blur = blur/256;
		blur = min(255, blur);
		int pindex = index(col, row);

		float red = gaussian_output[3 * pindex] / 256.0;
		float green = gaussian_output[(3 * pindex) + 1] / 256.0;
		float blue = gaussian_output[(3 * pindex) + 2] / 256.0;
		
		gaussian_output[3*pindex] = (unsigned char) (((float)blur)*red);
		gaussian_output[3*pindex+1] = (unsigned char) (((float)blur)*green);
		gaussian_output[3*pindex+2] = (unsigned char) (((float)blur)*blue);
		
        //gaussian_output[col + row * WIDTH] = static_cast<unsigned char>(blur);
    }
}

__global__
void render(int* sobolBuffer, unsigned char* frame, unsigned short time) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;

	float red = frame[3 * index] / 256.0;
	float green = frame[(3 * index) + 1] / 256.0;
	float blue = frame[(3 * index) + 2] / 256.0;

	int xv = sobolBuffer[2 * index];
	int yv = sobolBuffer[2 * index + 1];
	int mag = (int)sqrt((double) xv * xv + yv * yv);
	if (mag > 255)
		mag = 255;

	frame[3 * index] = red * mag;
	frame[(3 * index) + 1] = green * mag;
	frame[(3 * index) + 2] = blue * mag;
}

void filter(unsigned char* frame) {
	static unsigned short count = 0;
	dim3 thread(32, 32);
	dim3 block(WIDTH/32 + 1, HEIGHT/32 + 1);
	//dim3 block(40, 23);

	hipMemcpy(finished, frame, SIZE, hipMemcpyHostToDevice);
	greyScale<<<block, thread>>>(finished, greyScaleBuffer); //Convert to grayscale
	//printf("%d: %s\n", __LINE__, hipGetErrorString(hipGetLastError()));

	/* sobel */
	// sobelOp<<<block, thread>>>(greyScaleBuffer, sobel); //Compute Sobel convolution
	// render << <block, thread >> > (sobel, finished, count++);

	/* gaussian */
	gaussian_filter<<<block, thread>>>(greyScaleBuffer, finished); //Compute Sobel convolution	
	//printf("%d: %s\n", __LINE__, hipGetErrorString(hipGetLastError()));
	//render << <block, thread >> > (gaussian, finished, count++);

	count &= 0x3FF;

	hipDeviceSynchronize();
	hipMemcpy(frame, finished, SIZE, hipMemcpyDeviceToHost);
}


#include <stdio.h>
void initCuda() {
	hipMalloc(&greyScaleBuffer, WIDTH * HEIGHT);
	hipMalloc(&sobel, sizeof(int) * WIDTH * HEIGHT * 2);
	hipMalloc(&gaussian, sizeof(unsigned char) * WIDTH * HEIGHT * 3);
	hipMalloc(&finished, SIZE);
}

void freeCuda() {
	hipFree(greyScaleBuffer);
	hipFree(sobel);
	hipFree(gaussian);
	hipFree(finished);
}
