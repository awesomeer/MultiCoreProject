#include "hip/hip_runtime.h"
#include "../inc/kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <stdio.h>


/*
* This code is configured for a GTX 1060M with 1024 CUDA cores
*/


#define HEIGHT 720
#define WIDTH 1280
#define SIZE (3 * WIDTH * HEIGHT)


__managed__ char GX[9] = { 1, 0, -1,
						  2, 0, -2,
							1, 0, -1 };
__managed__ char GY[9] = { 1, 2, 1,
				   0, 0, 0,
				  -1,-2,-1 };

__managed__ char gaussian_kernel[9] = { 
	1/16, 1/8, 1/16,
	1/8, 1/4, 1/8,
	1/16, 1/8, 1/16,
};


unsigned char* greyScaleBuffer;
int *sobel; //1280x720
int *gaussian;
unsigned char *finished; //1280x720*3


__global__
void greyScale(unsigned char * frame, unsigned char*greyBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;
	int sum = (frame[3*index] + frame[3*index + 1] + frame[3*index + 2]) / 3;
	greyBuffer[index] = sum;
}


__device__
int index(int x, int y) {
	if (x >= WIDTH || y >= HEIGHT || x < 0 || y < 0)
		return -1;

	return x + y * WIDTH;
}

__device__ __forceinline__
int wrap(int val, int limit) {
	if (val < 0)
		return limit - 1;
	return val % limit;
}

__global__
void sobelOp(unsigned char * greyBuffer, int * sobelBuffer) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int xDir = 0;
	int yDir = 0;

	for (int r = -1; r < 2; r++) {
		int row = y + r;
		row = wrap(row, HEIGHT);

		for (int c = -1; c < 2; c++) {
			int col = x + c;
			col = wrap(col, WIDTH);

			xDir += greyBuffer[index(col, row)] * GX[(1 - c) + (1 - r) * 3];
			yDir += greyBuffer[index(col, row)] * GY[(1 - c) + (1 - r) * 3];
		}
	}

	__syncthreads();

	sobelBuffer[2*index(x, y)] = xDir;
	sobelBuffer[2*index(x, y)+1] = yDir;

}

__global__ void gaussian_filter(const unsigned char *gaussian_input, int *gaussian_output) {

    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

    if(row < HEIGHT && col < WIDTH) {
        float blur = 0.0;
        for(int i = -1; i < 2; i++) {
            for(int j = -1; j < 2; j++) {

                const unsigned int y = max(0, min(HEIGHT - 1, row + i));
                const unsigned int x = max(0, min(WIDTH - 1, col + j));

                const float w = gaussian_kernel[(j + 1) + (i + 1) * 3];
                blur += w * gaussian_input[x + y * WIDTH];
            }
        }
        gaussian_output[col + row * WIDTH] = static_cast<unsigned char>(blur);
    }
}

__global__
void render(int* sobolBuffer, unsigned char* frame, unsigned short time) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int index = x + y * WIDTH;

	float red = frame[3 * index] / 256.0;
	float green = frame[(3 * index) + 1] / 256.0;
	float blue = frame[(3 * index) + 2] / 256.0;

	int xv = sobolBuffer[2 * index];
	int yv = sobolBuffer[2 * index + 1];
	int mag = (int)sqrt((double) xv * xv + yv * yv);
	if (mag > 255)
		mag = 255;

	frame[3 * index] = red * mag;
	frame[(3 * index) + 1] = green * mag;
	frame[(3 * index) + 2] = blue * mag;
}

void filter(unsigned char* frame) {
	static unsigned short count = 0;
	dim3 thread(32, 32);
	dim3 block(WIDTH/32 + 1, HEIGHT/32 + 1);
	//dim3 block(40, 23);

	hipMemcpy(finished, frame, SIZE, hipMemcpyHostToDevice);
	greyScale<<<block, thread>>>(finished, greyScaleBuffer); //Convert to grayscale

	/* sobel */
	// sobelOp<<<block, thread>>>(greyScaleBuffer, sobel); //Compute Sobel convolution
	// render << <block, thread >> > (sobel, finished, count++);

	/* gaussian */
	gaussian_filter<<<block, thread>>>(greyScaleBuffer, gaussian); //Compute Sobel convolution	
	render << <block, thread >> > (gaussian, finished, count++);

	count &= 0x3FF;

	hipDeviceSynchronize();
	hipMemcpy(frame, finished, SIZE, hipMemcpyDeviceToHost);
}


#include <stdio.h>
void initCuda() {
	hipMalloc(&greyScaleBuffer, WIDTH * HEIGHT);
	hipMalloc(&sobel, sizeof(int) * WIDTH * HEIGHT * 2);
	hipMalloc(&gaussian, sizeof(int) * WIDTH * HEIGHT * 2);
	hipMalloc(&finished, SIZE);
}

void freeCuda() {
	hipFree(greyScaleBuffer);
	hipFree(sobel);
	hipFree(gaussian);
	hipFree(finished);
}